#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
extern "C" {
  #include "png_util.h"
}

__global__ void mandelbrotKernel(const int NRe, 
		const int NIm, 
		const float xmin,
		const float ymin,
		const float dRe, 
		const float dIm,
		float * h_count){

  // orient thread
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int dx = blockDim.x;
  
  int ty = threadIdx.y;
  int by = blockIdx.y;
  int dy = blockDim.y;
  
  // establish which entry this thread is responsible for
  int column = tx + bx*dx;
  int row    = ty + by*dy;

  if (row < NIm && column < NRe) { // bounds check
      float cRe = xmin + column*dRe;
      float cIm = ymin + row*dIm;

      float zRe = 0;
      float zIm = 0;
      
      int Nt = 200;
      int t, cnt=0;
      for(t=0;t<Nt;++t){
	
	// z = z^2 + c
	//   = (zRe + i*zIm)*(zRe + i*zIm) + (cRe + i*cIm)
	//   = zRe^2 - zIm^2 + 2*i*zIm*zRe + cRe + i*cIm
	      float zReTmp = zRe*zRe - zIm*zIm + cRe;
	      zIm = 2.f*zIm*zRe + cIm;
	      zRe = zReTmp;

	      cnt += (zRe*zRe+zIm*zIm<4.f);
      }

      h_count[column + row*NRe] = cnt;
  }

}


int main(int argc, char **argv){

  const int NRe = 1000;
  const int NIm = 1000;

  const float xmin = -2;
  const float xmax = 1; 
  const float ymin = -1.5; 
  const float ymax = 1.5;

  const float dRe = (xmax-xmin)/(NRe-1.f);
  const float dIm = (ymax-ymin)/(NIm-1.f);

  // allocate HOST array
  float *h_count = (float*) calloc(NRe*NIm, sizeof(float));

  // allocate DEVICE array
  float* c_count;
  hipMalloc(&c_count, NRe*NIm * sizeof(float));

  // establish threading dimensions
  dim3 B(16,16,1);
  dim3 G((NIm+16-1)/16, (NRe+16-1)/16, 1);

  // warm up kernel
  mandelbrotKernel <<< G , B >>> (NRe, NIm, xmin, ymin, dRe, dIm, c_count);

  // set up timing
  hipEvent_t tic, toc;
  hipEventCreate(&tic);
  hipEventCreate(&toc);

  // start timer
  hipDeviceSynchronize();
  hipEventRecord(tic);

  // call mandelbrot from here
  mandelbrotKernel <<< G , B >>> (NRe, NIm, xmin, ymin, dRe, dIm, c_count);

  // end timer
  hipEventRecord(toc);
  hipDeviceSynchronize();

  // get and print elapsed time
  float elapsed;
  hipEventElapsedTime(&elapsed, tic, toc);
  elapsed /= 1000;
  
  printf("Elapsed Time %f seconds\n", elapsed);

  // copy results from DEVICE to HOST
  hipMemcpy(h_count, c_count, NRe*NIm * sizeof(float), hipMemcpyDeviceToHost);

  // create the png
  FILE *png = fopen("mandelbrot.png", "w");
  write_hot_png(png, NRe, NIm, h_count, 0, 100);
  fclose(png);

}